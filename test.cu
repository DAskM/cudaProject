// #include "device_launch_parameters.h"
// #include <iostream>
// #include <cuda_runtime_api.h>

// int main()
// {
//     int deviceCount;
//     cudaGetDeviceCount(&deviceCount);
//     for(int i=0;i<deviceCount;i++)
//     {
//         cudaDeviceProp devProp;
//         cudaGetDeviceProperties(&devProp, i);
//         std::cout << "使用GPU device " << i << ": " << devProp.name << std::endl;
//         std::cout << "设备全局内存总量： " << devProp.totalGlobalMem / 1024 / 1024 << "MB" << std::endl;
//         std::cout << "SM的数量:" << devProp.multiProcessorCount << std::endl;
//         std::cout << "每个线程块的共享内存大小：" << devProp.sharedMemPerBlock / 1024.0 << " KB" << std::endl;
//         std::cout << "每个线程块的最大线程数：" << devProp.maxThreadsPerBlock << std::endl;
//         std::cout << "设备上一个线程块(Block)种可用的32位寄存器数量: " << devProp.regsPerBlock << std::endl;
//         std::cout << "每个EM的最大线程数:" << devProp.maxThreadsPerMultiProcessor << std::endl;
//         std::cout << "每个EM的最大线程束数:" << devProp.maxThreadsPerMultiProcessor / 32 << std::endl;
//         std::cout << "设备上多处理器的数量： " << devProp.multiProcessorCount << std::endl;
//         std::cout << "======================================================" << std::endl;     
//     }
//     return 0;
// }

#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
// Include CUDA runtime and CUFFT
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

// Helper functions for CUDA


#define pi 3.1415926535
#define LENGTH 100 //signal sampling points 采样点数
int main()
{
    // data gen 生成测试信号
    float Data[LENGTH] = { 1,2,3,4 };
    float fs = 1000000.000;//sampling frequency
    float f0 = 200000.00;// signal frequency
    for (int i = 0; i < LENGTH; i++)
    {
        Data[i] = 1.35*cos(2 * pi*f0*i / fs);//signal gen,
    }

    hipfftComplex *CompData = (hipfftComplex*)malloc(LENGTH * sizeof(hipfftComplex));//allocate memory for the data in host
    int i;
    for (i = 0; i < LENGTH; i++)
    {
        CompData[i].x = Data[i];
        CompData[i].y = 0;
    }

    hipfftComplex *d_fftData;
    hipMalloc((void**)&d_fftData, LENGTH * sizeof(hipfftComplex));// allocate memory for the data in device
    hipMemcpy(d_fftData, CompData, LENGTH * sizeof(hipfftComplex), hipMemcpyHostToDevice);// copy data from host to device

    hipfftHandle plan;// cuda library function handle
    hipfftPlan1d(&plan, LENGTH, HIPFFT_C2C, 1);//declaration
    hipfftExecC2C(plan, (hipfftComplex*)d_fftData, (hipfftComplex*)d_fftData, HIPFFT_FORWARD);//execute
    hipDeviceSynchronize();//wait to be done
    hipMemcpy(CompData, d_fftData, LENGTH * sizeof(hipfftComplex), hipMemcpyDeviceToHost);// copy the result from device to host

    for (i = 0; i < LENGTH / 2; i++)
    {
        printf("i=%d\tf= %6.1fHz\tRealAmp=%3.1f\t", i, fs*i / LENGTH, CompData[i].x*2.0 / LENGTH);
        printf("ImagAmp=+%3.1fi", CompData[i].y*2.0 / LENGTH);
        printf("\n");
    }
    hipfftDestroy(plan);
    free(CompData);
    hipFree(d_fftData);

}