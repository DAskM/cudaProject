#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <hip/hip_runtime_api.h>

#include "gpuInfoPrint.cuh"

void getGpuInfo(){
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    for(int i=0; i<deviceCount; i++){
        hipDeviceProp_t devProp;
        hipGetDeviceProperties(&devProp, i);
        std::cout << "GPU device " << i << ": " << devProp.name << std::endl;
        std::cout << "total memory: " << devProp.totalGlobalMem /1024 /1024 << "MB" << std::endl;
        std::cout << "SM number: " << devProp.multiProcessorCount << std::endl;
        std::cout << "shared memoty size for each thread: " << devProp.sharedMemPerBlock << std::endl;
        std::cout << "max thread per block: " << devProp.maxThreadsPerBlock << std::endl; 
    }
    
}